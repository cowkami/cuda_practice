
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10000000000

__global__ void add_gpu(float *a, float *b, float *c) {
	long long tid = blockIdx.x;
	if (tid < N) 
		c[tid] = a[tid] + b[tid];
}

int main(void) {
	float a[N], b[N], c[N];
	float *dev_a, *dev_b, *dev_c;

	hipMalloc((void**)&dev_a, N * sizeof(float));
	hipMalloc((void**)&dev_b, N * sizeof(float));
	hipMalloc((void**)&dev_c, N * sizeof(float));

	for (long long i=0; i<N; i++) {
		a[i] = -i;
		b[i] = i*3;
	}

	hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice);

	add_gpu<<<N, 1>>>(dev_a, dev_b, dev_c);

	hipMemcpy(c, dev_c, N * sizeof(float), hipMemcpyDeviceToHost);

	// for (int i=0; i<N; i++) {
	// 	printf("%d + %d = %d\n", a[i], b[i], c[i]);
	// }

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}
